#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

#include <cmath>

#define BLOCK_SIZE 256

extern "C" __global__ void Gaussian_kernel_density_estimate(float* X, float* Y, int N, float h) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        float sum = 0.0f;
        for (int i = 0; i < N; ++i) {
            float diff = X[index] - X[i];
            float weight = exp(-diff * diff / (2.0f * h * h));
            sum += weight;
        }
        Y[index] = sum / (N * h * sqrtf(2.0f * 3.141592654f));
    }
}