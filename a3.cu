#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>

#define BLOCK_SIZE 256

extern "C" __global__ void Gaussian_kernel_density_estimate(float* X, float* Y, int N, float h) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        float sum = 0.0f;
        for (int i = 0; i < N; ++i) {
            float diff = X[index] - X[i];
            float weight = exp(-diff * diff / (2.0f * h * h));
            sum += weight;
        }
        Y[index] = sum / (N * h * sqrtf(2.0f * 3.141592654f));
    }
}

void wrapper(int n, float h, const std::vector<float>& x, std::vector<float>& y) {
    float* X_d, * Y_d;

    hipMalloc(&X_d, n * sizeof(float));
    hipMalloc(&Y_d, n * sizeof(float));

    // Copy data from host to device
    hipMemcpy(X_d, x.data(), n * sizeof(float), hipMemcpyHostToDevice);

    int num_blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    Gaussian_kernel_density_estimate<<<num_blocks, BLOCK_SIZE>>>(X_d, Y_d, n, h);

    // Copy data from device to host
    hipMemcpy(y.data(), Y_d, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(X_d);
    hipFree(Y_d);
}